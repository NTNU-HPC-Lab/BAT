#include "hip/hip_runtime.h"
#if PRECISION == 32 
    // Texture Readers
    texture<float, 1> vecTex;    

    struct texReader {
        __device__ __forceinline__ float operator()(const int idx) const
        {
            return tex1D(vecTex, idx);
        }
    };
#elif PRECISION == 64
    // Texture Readers
    texture<float, 1> vecTex;
    struct texReader {
        __device__ __forceinline__ double operator()(const int idx) const
        {
            return tex1D(vecTex, idx);
        }
     };
#endif


// ****************************************************************************
// Function: spmv_csr_scalar_kernel
//
// Purpose:
//   Computes sparse matrix - vector multiplication on the GPU using
//   the CSR data storage format, using a thread per row of the sparse
//   matrix; based on Bell (SC09) and Baskaran (IBM Tech Report)
//
// Arguments:
//   val: array holding the non-zero values for the matrix
//   cols: array of column indices for each element of the sparse matrix
//   rowDelimiters: array of size dim+1 holding indices to rows of the matrix
//                  last element is the index one past the last
//                  element of the matrix
//   dim: number of rows in the matrix
//   out: output - result from the spmv calculation
//
// Returns:  nothing
//           out indirectly through a pointer
//
// Programmer: Lukasz Wesolowski
// Creation: June 28, 2010
//
// Modifications:
//
// ****************************************************************************
// template <typename fpType, typename texReader>
extern "C" __global__ void
spmv_csr_scalar_kernel( float * valSP,
                         double * valDP,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, float * outSP,
                       double * outDP)
{
    int myRow = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    texReader vecTexReader;

    if (myRow < dim)
    {
        #if PRECISION == 32
            float t = 0.0f;
        #else 
            double t = 0.0;
        #endif
        int start = rowDelimiters[myRow];
        int end = rowDelimiters[myRow+1];
        #if UNROLL_LOOP
        #pragma unroll
        #endif

        for (int j = start; j < end; j++)
        {
            int col = cols[j];

            #if PRECISION == 32
                t += valSP[j] * vecTexReader(col);
            #else 
                t += valDP[j]* vecTexReader(col);
            #endif
        }
        #if PRECISION == 32
            outSP[myRow] = t;
        #else 
            outDP[myRow] = t;
        #endif
        
    }
}

// ****************************************************************************
// Function: spmv_csr_vector_kernel
//
// Purpose:
//   Computes sparse matrix - vector multiplication on the GPU using
//   the CSR data storage format, using a warp per row of the sparse
//   matrix; based on Bell (SC09) and Baskaran (IBM Tech Report)
//
// Arguments:
//   val: array holding the non-zero values for the matrix
//   cols: array of column indices for each element of the sparse matrix
//   rowDelimiters: array of size dim+1 holding indices to rows of the matrix
//                  last element is the index one past the last
//                  element of the matrix
//   dim: number of rows in the matrix
//   out: output - result from the spmv calculation
//
// Returns:  nothing
//           out indirectly through a pointer
//
// Programmer: Lukasz Wesolowski
// Creation: June 28, 2010
//
// Modifications:
//
// ****************************************************************************
template <typename fpType, typename texReader>
__global__ void
spmv_csr_vector_kernel(const fpType * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, fpType * __restrict__ out)
{
    // Thread ID in block
    int t = threadIdx.x;
    // Thread ID within warp
    int id = t & (warpSize-1);
    int warpsPerBlock = blockDim.x / warpSize;
    // One row per warp
    int myRow = (blockIdx.x * warpsPerBlock) + (t / warpSize);
    // Texture reader for the dense vector
    texReader vecTexReader;

    __shared__ volatile fpType partialSums[BLOCK_SIZE];

    if (myRow < dim) {
        int warpStart = rowDelimiters[myRow];
        int warpEnd = rowDelimiters[myRow+1];
        fpType mySum = 0;
        #if UNROLL_LOOP
        #pragma unroll
        #endif
        for (int j = warpStart + id; j < warpEnd; j += warpSize)
        {
            int col = cols[j];
            mySum += val[j] * vecTexReader(col);
        }
        partialSums[t] = mySum;

        // Reduce partial sums
        if (id < 16) {
            #if UNROLL_LOOP_2
            #pragma unroll
            #endif
            for (int i = 4; i >= 0; i--) {
                int l = 1<<i;
                if (id < l) partialSums[t] += partialSums[t+l];
            }
        }
        
        // Write result
        if (id == 0) {
            out[myRow] = partialSums[t];
        }
    }
}

// ****************************************************************************
// Function: spmv_ellpackr_kernel
//
// Purpose:
//   Computes sparse matrix - vector multiplication on the GPU using
//   the ELLPACK-R data storage format; based on Vazquez et al (Univ. of
//   Almeria Tech Report 2009)
//
// Arguments:
//   val: array holding the non-zero values for the matrix in column
//   major format and padded with zeros up to the length of longest row
//   cols: array of column indices for each element of the sparse matrix
//   rowLengths: array storing the length of each row of the sparse matrix
//   dim: number of rows in the matrix
//   out: output - result from the spmv calculation
//
// Returns:  nothing directly
//           out indirectly through a pointer
//
// Programmer: Lukasz Wesolowski
// Creation: June 29, 2010
//
// Modifications:
//
// ****************************************************************************
template <typename fpType, typename texReader>
__global__ void
spmv_ellpackr_kernel(const fpType * __restrict__ val,
                     const int    * __restrict__ cols,
                     const int    * __restrict__ rowLengths,
                     const int dim, fpType * __restrict__ out)
{
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    texReader vecTexReader;

    if (t < dim)
    {
        fpType result = 0.0f;
        int max = rowLengths[t];

        #if UNROLL_LOOP
        #pragma unroll
        #endif
        for (int i = 0; i < max; i++)
        {
            int ind = i*dim+t;
            result += val[ind] * vecTexReader(cols[ind]);
        }
        out[t] = result;
    }
}

