#include "hip/hip_runtime.h"
template <typename fpType, typename texReader>
__device__ void
spmv_csr_scalar_kernel(const fpType * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, fpType * __restrict__ out);

template <typename fpType, typename texReader>
__device__ void
spmv_csr_vector_kernel(const fpType * __restrict__ val,
                        const int    * __restrict__ cols,
                        const int    * __restrict__ rowDelimiters,
                        const int dim, fpType * __restrict__ out);

template <typename fpType, typename texReader>
__device__ void
spmv_ellpackr_kernel(const fpType * __restrict__ val,
                        const int    * __restrict__ cols,
                        const int    * __restrict__ rowLengths,
                        const int dim, fpType * __restrict__ out);

#if PRECISION == 32 
    // Texture Readers
    texture<float, 1> vecTex;    

    struct texReader {
        __device__ __forceinline__ float operator()(const int idx) const
        {
            return tex1D(vecTex, idx);
        }
    };
#elif PRECISION == 64
    // Texture Readers
    texture<float, 1> vecTex;
    struct texReader {
        __device__ __forceinline__ double operator()(const int idx) const
        {
            return tex1D(vecTex, idx);
        }
     };
#endif



/**
 * Helper function for tuners that can not use templated kernels directly
 * This function also chooses format based on a parameter
 */
extern "C" __global__ void
spmv_kernel(float * valSP_csr,
            double * valDP_csr,
            float * valSP_ellpackr,
            double * valDP_ellpackr,
            const int    * __restrict__ cols_csr,
            const int    * __restrict__ cols_ellpackr,
            const int    * __restrict__ rowDelimiters,
            const int    * __restrict__ rowLengths,
            const int dim, 
            float * outSP_csr,
            double * outDP_csr,
            float * outSP_ellpackr,
            double * outDP_ellpackr) {

    #if PRECISION == 32
        #if (FORMAT == 1 || FORMAT == 2)
            spmv_csr_scalar_kernel<float, texReader>(valSP_csr, cols_csr, rowDelimiters, dim, outSP_csr);
        #elif (FORMAT == 3 || FORMAT == 4)
            spmv_csr_vector_kernel<float, texReader>(valSP_csr, cols_csr, rowDelimiters, dim, outSP_csr);
        #else
            spmv_ellpackr_kernel<float, texReader>(valSP_ellpackr, cols_ellpackr, rowLengths, dim, outSP_ellpackr);
        #endif
    #else 
        #if (FORMAT == 1 || FORMAT == 2)
            spmv_csr_scalar_kernel<double, texReader>(valDP_csr, cols_csr, rowDelimiters, dim, outDP_csr);
        #elif (FORMAT == 3 || FORMAT == 4)
            spmv_csr_vector_kernel<double, texReader>(valDP_csr, cols_csr, rowDelimiters, dim, outDP_csr);
        #else
            spmv_ellpackr_kernel<double, texReader>(valDP_ellpackr, cols_ellpackr, rowLengths, dim, outDP_ellpackr);
        #endif
    #endif
}

// ****************************************************************************
// Function: spmv_csr_scalar_kernel
//
// Purpose:
//   Computes sparse matrix - vector multiplication on the GPU using
//   the CSR data storage format, using a thread per row of the sparse
//   matrix; based on Bell (SC09) and Baskaran (IBM Tech Report)
//
// Arguments:
//   val: array holding the non-zero values for the matrix
//   cols: array of column indices for each element of the sparse matrix
//   rowDelimiters: array of size dim+1 holding indices to rows of the matrix
//                  last element is the index one past the last
//                  element of the matrix
//   dim: number of rows in the matrix
//   out: output - result from the spmv calculation
//
// Returns:  nothing
//           out indirectly through a pointer
//
// Programmer: Lukasz Wesolowski
// Creation: June 28, 2010
//
// Modifications:
//
// ****************************************************************************
template <typename fpType, typename texReader>
__device__ void
spmv_csr_scalar_kernel(const fpType * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, fpType * __restrict__ out)
{
    int myRow = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    texReader vecTexReader;

    if (myRow < dim)
    {
        fpType t = 0.0f;
        int start = rowDelimiters[myRow];
        int end = rowDelimiters[myRow+1];
        #if UNROLL_LOOP_1
        #pragma unroll
        #endif
        for (int j = start; j < end; j++)
        {
            int col = cols[j];
            t += val[j] * vecTexReader(col);
        }
        out[myRow] = t;
    }
}

// ****************************************************************************
// Function: spmv_csr_vector_kernel
//
// Purpose:
//   Computes sparse matrix - vector multiplication on the GPU using
//   the CSR data storage format, using a warp per row of the sparse
//   matrix; based on Bell (SC09) and Baskaran (IBM Tech Report)
//
// Arguments:
//   val: array holding the non-zero values for the matrix
//   cols: array of column indices for each element of the sparse matrix
//   rowDelimiters: array of size dim+1 holding indices to rows of the matrix
//                  last element is the index one past the last
//                  element of the matrix
//   dim: number of rows in the matrix
//   out: output - result from the spmv calculation
//
// Returns:  nothing
//           out indirectly through a pointer
//
// Programmer: Lukasz Wesolowski
// Creation: June 28, 2010
//
// Modifications:
//
// ****************************************************************************
template <typename fpType, typename texReader>
__device__ void
spmv_csr_vector_kernel(const fpType * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, fpType * __restrict__ out)
{
    // Thread ID in block
    int t = threadIdx.x;
    // Thread ID within warp
    int id = t & (warpSize-1);
    int warpsPerBlock = BLOCK_SIZE / warpSize;
    // One row per warp
    int myRow = (blockIdx.x * warpsPerBlock) + (t / warpSize);
    // Texture reader for the dense vector
    texReader vecTexReader;

    __shared__ volatile fpType partialSums[BLOCK_SIZE];

    if (myRow < dim) {
        int warpStart = rowDelimiters[myRow];
        int warpEnd = rowDelimiters[myRow+1];
        fpType mySum = 0;
        #if UNROLL_LOOP_1
        #pragma unroll
        #endif
        for (int j = warpStart + id; j < warpEnd; j += warpSize)
        {
            int col = cols[j];
            mySum += val[j] * vecTexReader(col);
        }
        partialSums[t] = mySum;

        // Reduce partial sums
        if (id < 16) {
            #if UNROLL_LOOP_2
            #pragma unroll
            #endif
            for (int i = 4; i >= 0; i--) {
                int l = 1<<i;
                if (id < l) partialSums[t] += partialSums[t+l];
            }
        }
        
        // Write result
        if (id == 0) {
            out[myRow] = partialSums[t];
        }
    }
}

// ****************************************************************************
// Function: spmv_ellpackr_kernel
//
// Purpose:
//   Computes sparse matrix - vector multiplication on the GPU using
//   the ELLPACK-R data storage format; based on Vazquez et al (Univ. of
//   Almeria Tech Report 2009)
//
// Arguments:
//   val: array holding the non-zero values for the matrix in column
//   major format and padded with zeros up to the length of longest row
//   cols: array of column indices for each element of the sparse matrix
//   rowLengths: array storing the length of each row of the sparse matrix
//   dim: number of rows in the matrix
//   out: output - result from the spmv calculation
//
// Returns:  nothing directly
//           out indirectly through a pointer
//
// Programmer: Lukasz Wesolowski
// Creation: June 29, 2010
//
// Modifications:
//
// ****************************************************************************
template <typename fpType, typename texReader>
__device__ void
spmv_ellpackr_kernel(const fpType * __restrict__ val,
                     const int    * __restrict__ cols,
                     const int    * __restrict__ rowLengths,
                     const int dim, fpType * __restrict__ out)
{
    int t = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    texReader vecTexReader;

    if (t < dim)
    {
        fpType result = 0.0f;
        int max = rowLengths[t];

        #if UNROLL_LOOP_1
        #pragma unroll
        #endif
        for (int i = 0; i < max; i++)
        {
            int ind = i*dim+t;
            result += val[ind] * vecTexReader(cols[ind]);
        }
        out[t] = result;
    }
}

