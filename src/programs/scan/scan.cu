#include "hip/hip_runtime.h"
#include "cudacommon.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>

#include <cassert>
#include <iostream>
#include <vector>

#include "OptionParser.h"
#include "scan.h"
#include "scan_kernel.h"

using namespace std;


// ****************************************************************************
// Function: addBenchmarkSpecOptions
//
// Purpose:
//   Add benchmark specific options parsing
//
// Arguments:
//   op: the options parser / parameter database
//
// Returns:  nothing
//
// Programmer: Kyle Spafford
// Creation: August 13, 2009
//
// Modifications:
//
// ****************************************************************************
void addBenchmarkSpecOptions(OptionParser &op)
{
    op.addOption("iterations", OPT_INT, "256", "specify scan iterations");
}

// ****************************************************************************
// Function: RunBenchmark
//
// Purpose:
//   Executes the scan (parallel prefix sum) benchmark
//
// Arguments:
//   resultDB: results from the benchmark are stored in this db
//   op: the options parser / parameter database
//
// Returns:  nothing
//
// Programmer: Kyle Spafford
// Creation: August 13, 2009
//
// Modifications:
// 5/18/2011 - KS - Changing to a non-recursive algorithm
// ****************************************************************************
void
RunBenchmark(OptionParser &op) {
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    #if PRECISION == 32
        cout << "Running single precision test" << endl;
        RunTest<float, float4>("Scan", op);
    #else
        cout << "Running double precision test" << endl;
        RunTest<double, double4>("Scan-DP", op);
    #endif
}

template <class T, class vecT>
void RunTest(string testName, OptionParser &op)
{
    int probSizes[4] = { 1, 8, 32, 64 };

    int size = probSizes[op.getOptionInt("size")-1];

    // Convert to MiB
    size = (size * 1024 * 1024) / sizeof(T);
    
    // create input data on CPU
    unsigned int bytes = size * sizeof(T);
    cout << size << endl;
    cout << bytes << endl;

    // Allocate Host Memory
    T* h_idata;
    T* reference;
    T* h_odata;
    CUDA_SAFE_CALL(hipHostMalloc((void**) &h_idata,   bytes));
    CUDA_SAFE_CALL(hipHostMalloc((void**) &reference, bytes));
    CUDA_SAFE_CALL(hipHostMalloc((void**) &h_odata,   bytes));

    // Initialize host memory
    cout << "Initializing host memory." << endl;
    for (int i = 0; i < size; i++)
    {
        h_idata[i] = i % 2; // Fill with some pattern
        h_odata[i] = -1;
    }

    // Thread configuration
    // Note: changing this may require updating the kernel calls below
    int num_blocks  = GRID_SIZE;//BLOCK_SIZE/GRID_SIZE_PART;
    int num_threads = BLOCK_SIZE;

    int smem_size = sizeof(T) * num_threads;

    // Allocate device memory
    T* d_idata, *d_odata, *d_block_sums;
    CUDA_SAFE_CALL(hipMalloc((void**) &d_idata, bytes));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_odata, bytes));
    CUDA_SAFE_CALL(hipMalloc((void**) &d_block_sums, num_blocks * sizeof(T)));

    // Copy data to GPU
    cout << "Copying data to device." << endl;
    CUDA_SAFE_CALL(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));

    int passes = op.getOptionInt("passes");
    int iters = op.getOptionInt("iterations");

    cout << "Running benchmark with size " << size << endl;
    for (int k = 0; k < passes; k++) {
        for (int j = 0; j < iters; j++) {
            // For scan, we use a reduce-then-scan approach

            // Each thread block gets an equal portion of the
            // input array, and computes the sum.
            reduce<T, BLOCK_SIZE><<<num_blocks, num_threads, smem_size>>>
                (d_idata, d_block_sums, size);

            // Next, a top-level exclusive scan is performed on the array
            // of block sums
            scan_single_block<T, BLOCK_SIZE><<<1, num_threads, smem_size*2>>>
                (d_block_sums, num_blocks);

            // Finally, a bottom-level scan is performed by each block
            // that is seeded with the scanned value in block sums
            bottom_scan<T, vecT, BLOCK_SIZE><<<num_blocks, num_threads, 2*smem_size>>>
                (d_idata, d_odata, d_block_sums, size);
        }
        CUDA_SAFE_CALL(hipMemcpy(h_odata, d_odata, bytes,
                hipMemcpyDeviceToHost));

        // If results aren't correct, don't report perf numbers
        if (! scanCPU<T>(h_idata, reference, h_odata, size))
        {
            return;
        }
    }
    CUDA_SAFE_CALL(hipFree(d_idata));
    CUDA_SAFE_CALL(hipFree(d_odata));
    CUDA_SAFE_CALL(hipFree(d_block_sums));
    CUDA_SAFE_CALL(hipHostFree(h_idata));
    CUDA_SAFE_CALL(hipHostFree(h_odata));
    CUDA_SAFE_CALL(hipHostFree(reference));
}


// ****************************************************************************
// Function: scanCPU
//
// Purpose:
//   Simple cpu scan routine to verify device results
//
// Arguments:
//   data : the input data
//   reference : space for the cpu solution
//   dev_result : result from the device
//   size : number of elements
//
// Returns:  nothing, prints relevant info to stdout
//
// Programmer: Kyle Spafford
// Creation: August 13, 2009
//
// Modifications:
//
// ****************************************************************************
template <class T>
bool scanCPU(T *data, T* reference, T* dev_result, const size_t size)
{

    bool passed = true;
    T last = 0.0f;

    for (unsigned int i = 0; i < size; ++i)
    {
        reference[i] = data[i] + last;
        last = reference[i];
    }
    for (unsigned int i = 0; i < size; ++i)
    {
        if (reference[i] != dev_result[i])
        {
#ifdef VERBOSE_OUTPUT
            cout << "Mismatch at i: " << i << " ref: " << reference[i]
                 << " dev: " << dev_result[i] << endl;
#endif
            passed = false;
        }
    }
    cout << "Test ";
    if (passed)
        cout << "Passed" << endl;
    else
        cerr << "---FAILED---" << endl;
    return passed;
}
