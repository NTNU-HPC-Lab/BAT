#include "hip/hip_runtime.h"
#include "spmv.h"

// ****************************************************************************
// Function: spmv_csr_scalar_kernel
//
// Purpose:
//   Computes sparse matrix - vector multiplication on the GPU using
//   the CSR data storage format, using a thread per row of the sparse
//   matrix; based on Bell (SC09) and Baskaran (IBM Tech Report)
//
// Arguments:
//   val: array holding the non-zero values for the matrix
//   cols: array of column indices for each element of the sparse matrix
//   rowDelimiters: array of size dim+1 holding indices to rows of the matrix
//                  last element is the index one past the last
//                  element of the matrix
//   dim: number of rows in the matrix
//   out: output - result from the spmv calculation
//
// Returns:  nothing
//           out indirectly through a pointer
//
// Programmer: Lukasz Wesolowski
// Creation: June 28, 2010
//
// Modifications:
//
// ****************************************************************************
template <typename fpType, typename texReader>
__global__ void
spmv_csr_scalar_kernel(const fpType * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, fpType * __restrict__ out)
{
    int myRow = blockIdx.x * blockDim.x + threadIdx.x;
    texReader vecTexReader;

    if (myRow < dim)
    {
        fpType t = 0.0f;
        int start = rowDelimiters[myRow];
        int end = rowDelimiters[myRow+1];
        #if UNROLL_LOOP_1
        #pragma unroll
        #endif
        for (int j = start; j < end; j++)
        {
            int col = cols[j];
            t += val[j] * vecTexReader(col);
        }
        out[myRow] = t;
    }
}

// ****************************************************************************
// Function: spmv_csr_vector_kernel
//
// Purpose:
//   Computes sparse matrix - vector multiplication on the GPU using
//   the CSR data storage format, using a warp per row of the sparse
//   matrix; based on Bell (SC09) and Baskaran (IBM Tech Report)
//
// Arguments:
//   val: array holding the non-zero values for the matrix
//   cols: array of column indices for each element of the sparse matrix
//   rowDelimiters: array of size dim+1 holding indices to rows of the matrix
//                  last element is the index one past the last
//                  element of the matrix
//   dim: number of rows in the matrix
//   out: output - result from the spmv calculation
//
// Returns:  nothing
//           out indirectly through a pointer
//
// Programmer: Lukasz Wesolowski
// Creation: June 28, 2010
//
// Modifications:
//
// ****************************************************************************
template <typename fpType, typename texReader>
__global__ void
spmv_csr_vector_kernel(const fpType * __restrict__ val,
                       const int    * __restrict__ cols,
                       const int    * __restrict__ rowDelimiters,
                       const int dim, fpType * __restrict__ out)
{
    // Thread ID in block
    int t = threadIdx.x;
    // Thread ID within warp
    int id = t & (warpSize-1);
    int warpsPerBlock = blockDim.x / warpSize;
    // One row per warp
    int myRow = (blockIdx.x * warpsPerBlock) + (t / warpSize);
    // Texture reader for the dense vector
    texReader vecTexReader;

    __shared__ volatile fpType partialSums[BLOCK_SIZE];

    if (myRow < dim) {
        int warpStart = rowDelimiters[myRow];
        int warpEnd = rowDelimiters[myRow+1];
        fpType mySum = 0;
        #if UNROLL_LOOP_1
        #pragma unroll
        #else
        #pragma unroll(1)
        #endif
        for (int j = warpStart + id; j < warpEnd; j += warpSize)
        {
            int col = cols[j];
            mySum += val[j] * vecTexReader(col);
        }
        partialSums[t] = mySum;

        // Reduce partial sums
        if (id < 16) {
            #if UNROLL_LOOP_2
            #pragma unroll
            #else
            #pragma unroll(1)
            #endif
            for (int i = 4; i >= 0; i--) {
                int l = 1<<i;
                if (id < l) partialSums[t] += partialSums[t+l];
            }
        }
        
        // Write result
        if (id == 0) {
            out[myRow] = partialSums[t];
        }
    }
}

// ****************************************************************************
// Function: spmv_ellpackr_kernel
//
// Purpose:
//   Computes sparse matrix - vector multiplication on the GPU using
//   the ELLPACK-R data storage format; based on Vazquez et al (Univ. of
//   Almeria Tech Report 2009)
//
// Arguments:
//   val: array holding the non-zero values for the matrix in column
//   major format and padded with zeros up to the length of longest row
//   cols: array of column indices for each element of the sparse matrix
//   rowLengths: array storing the length of each row of the sparse matrix
//   dim: number of rows in the matrix
//   out: output - result from the spmv calculation
//
// Returns:  nothing directly
//           out indirectly through a pointer
//
// Programmer: Lukasz Wesolowski
// Creation: June 29, 2010
//
// Modifications:
//
// ****************************************************************************
template <typename fpType, typename texReader>
__global__ void
spmv_ellpackr_kernel(const fpType * __restrict__ val,
                     const int    * __restrict__ cols,
                     const int    * __restrict__ rowLengths,
                     const int dim, fpType * __restrict__ out)
{
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    texReader vecTexReader;

    if (t < dim)
    {
        fpType result = 0.0f;
        int max = rowLengths[t];

        #if UNROLL_LOOP_1
        #pragma unroll
        #else
        #pragma unroll(1)
        #endif
        for (int i = 0; i < max; i++)
        {
            int ind = i*dim+t;
            result += val[ind] * vecTexReader(cols[ind]);
        }
        out[t] = result;
    }
}

