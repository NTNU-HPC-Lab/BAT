#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <string.h>

#include <cassert>
#include <fstream>
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

extern "C" {

#include "reduction_kernel.cu"

using namespace std;

// Select which precision that are used in the calculations
// And define the replacements for the template inputs
#if PRECISION == 32
    #define T float
#elif PRECISION == 64
    #define T double
#endif

float RunTest(string testName);

// ****************************************************************************
// Function: reduceCPU
//
// Purpose:
//   Simple cpu reduce routine to verify device results
//
// Arguments:
//   data : the input data
//   size : size of the input data
//
// Returns:  sum of the data
//
// Programmer: Kyle Spafford
// Creation: August 13, 2009
//
// Modifications:
//
// ****************************************************************************
T reduceCPU(const T *data, int size)
{
    T sum = 0;
    for (int i = 0; i < size; i++)
    {
        sum += data[i];
    }
    return sum;
}

// ****************************************************************************
// Function: RunBenchmark
//
// Purpose:
//   Driver for the reduction benchmark.  Detects double precision capability
//   and calls the RunTest function appropriately
//
// Arguments:
//   resultDB: results from the benchmark are stored in this db
//   op: the options parser / parameter database
//
// Returns:  nothing
//
// Programmer: Kyle Spafford
// Creation: August 13, 2009
//
// Modifications:
//
// ****************************************************************************

// Return the time used in float to help choosing the best configuration
float reduction_host() {
    #if PRECISION == 32
        cout << "Running single precision test" << endl;
        return RunTest("Reduction");
    #elif PRECISION == 64
        cout << "Running double precision test" << endl;
        return RunTest("Reduction-DP");
    #endif
}

// ****************************************************************************
// Function: RunTest
//
// Purpose:
//   Primary method for the reduction benchmark
//
// Arguments:
//   testName: the name of the test currently being executed (specifying SP or
//             DP)
//   resultDB: results from the benchmark are stored in this db
//   op: the options parser / parameter database
//
// Returns:  nothing
//
// Programmer: Kyle Spafford
// Creation: August 13, 2009
//
// Modifications:
//
// ****************************************************************************
float RunTest(string testName)
{
    int prob_sizes[4] = { 1, 8, 32, 64 };

    int input_problem_size = PROBLEM_SIZE;
    int size = prob_sizes[input_problem_size - 1];
    size = (size * 1024 * 1024) / sizeof(T);

    T* h_idata;
    hipHostMalloc((void**)&h_idata, size * sizeof(T));

    // Initialize host memory
    cout << "Initializing host memory." << endl;
    for(int i = 0; i < size; i++)
    {
        h_idata[i] = i % 3; //Fill with some pattern
    }

    // allocate device memory
    T* d_idata;
    hipMalloc((void**)&d_idata, size * sizeof(T));

    // TODO: remove this NB comment???
    int num_threads = BLOCK_SIZE; // NB: Update template to kernel launch if this is changed
    int num_blocks = GRID_SIZE;
    int smem_size = sizeof(T) * num_threads;
    // allocate mem for the result on host side
    T* h_odata;
    hipHostMalloc((void**)&h_odata, num_blocks * sizeof(T));

    T* d_odata;
    hipMalloc((void**)&d_odata, num_blocks * sizeof(T));

    int passes = 10;
    int iters  = 100;

    // For measuring the time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float totalElapsedTime = 0.0;

    cout << "Running benchmark." << endl;
    for (int k = 0; k < passes; k++)
    {
        // Copy data to GPU
        hipMemcpy(d_idata, h_idata, size * sizeof(T), hipMemcpyHostToDevice);

        hipTextureObject_t idataTextureObject = 0;
        
    #if TEXTURE_MEMORY
        // Setup the texture memory
        // Create the texture resource descriptor
        hipResourceDesc resourceDescriptor;
        memset(&resourceDescriptor, 0, sizeof(resourceDescriptor));
        resourceDescriptor.resType = hipResourceTypeLinear;
        resourceDescriptor.res.linear.devPtr = d_idata;
        #if PRECISION == 32
            resourceDescriptor.res.linear.desc.f = hipChannelFormatKindFloat;
        #elif PRECISION == 64
            resourceDescriptor.res.linear.desc.f = hipChannelFormatKindUnsigned;
        #endif
        resourceDescriptor.res.linear.desc.x = 32;
        #if PRECISION == 64
            resourceDescriptor.res.linear.desc.y = 32;
        #endif
        resourceDescriptor.res.linear.sizeInBytes = size * sizeof(T);

        // Create the texture resource descriptor
        hipTextureDesc textureDescriptor;
        memset(&textureDescriptor, 0, sizeof(textureDescriptor));
        textureDescriptor.readMode = hipReadModeElementType;

        // Create the texture object
        hipCreateTextureObject(&idataTextureObject, &resourceDescriptor, &textureDescriptor, NULL);
    #endif

        // Start the timing
        hipEventRecord(start, 0);

        // Execute kernel
        for (int m = 0; m < iters; m++)
        {
            reduce<<<num_blocks,num_threads, smem_size>>>
                (d_idata, idataTextureObject, d_odata, size);
        }

        // Stop the events and save elapsed time
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, start, stop);
        totalElapsedTime += elapsedTime;

        // Copy back to host
        hipMemcpy(h_odata, d_odata, num_blocks * sizeof(T), hipMemcpyDeviceToHost);

        T dev_result = 0;
        for (int i=0; i<num_blocks; i++)
        {
            dev_result += h_odata[i];
        }

        // compute reference solution
        T cpu_result = reduceCPU(h_idata, size);
        double threshold = 1.0e-6;
        T diff = fabs(dev_result - cpu_result);

        cout << "Test ";
        if (diff < threshold) {
            cout << "Passed" << endl;
        } else {
            cout << "FAILED" << endl;
            cout << "Diff: " << diff << endl;
            cerr << "Error: incorrect computed result." << endl;
            // (don't report erroneous results)
            throw "Correctness verification failed";
        }
    }
    hipHostFree(h_idata);
    hipHostFree(h_odata);
    hipFree(d_idata);
    hipFree(d_odata);

    return totalElapsedTime;
}
}
