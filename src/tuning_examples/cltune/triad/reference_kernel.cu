#include "hip/hip_runtime.h"
#if PRECISION == 32
    #define DATA_TYPE float
#elif PRECISION == 64
    #define DATA_TYPE double
#endif

// ****************************************************************************
// Function: triad
//
// Purpose:
//   A simple vector addition kernel
//   C = A + s*B
//
// Arguments:
//   A,B - input vectors
//   C - output vectors
//   s - scalar
//
// Returns:  nothing
//
// Programmer: Kyle Spafford
// Creation: December 15, 2009
//
// Modifications:
//
// ****************************************************************************
extern "C" __global__ void triad(DATA_TYPE* A, DATA_TYPE* B, DATA_TYPE* C, DATA_TYPE s, int numberOfElements)
{
    int gid = threadIdx.x + (blockIdx.x * blockDim.x);
    
    // Ensure that the current thread id is less than total number of elements
    if (gid < numberOfElements) {
        C[gid] = A[gid] + s*B[gid];
    }
}

extern "C" __global__ void triad_helper(float* Af, float* Bf, float* Cf, float sf, double* Ad, double* Bd, double* Cd, double sd, int numberOfElements) {
    #if PRECISION == 32
        triad(Af, Bf, Cf, sf, numberOfElements);
    #elif PRECISION == 64
        triad(Ad, Bd, Cd, sd, numberOfElements);
    #endif
}