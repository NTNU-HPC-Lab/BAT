#include <iostream>
#include <hip/hip_runtime.h>
#include "stdio.h"

#define WIDTH 2048
#define block_size_x 32
#define block_size_y 8
#define tile_size_x 4
#define tile_size_y 4

// Forward declaration of the CUDA kernel
__global__ void matmul_kernel(float *C, float *A, float *B);

// Utility function to check for CUDA errors
void checkCudaErrors(hipError_t result) {
    if (result != hipSuccess) {
        std::cerr << "CUDA Runtime Error: " << hipGetErrorString(result) << std::endl;
        exit(-1);
    }
}

// Main program
int main() {
    size_t size = WIDTH * WIDTH * sizeof(float);

    // Allocate and initialize matrices A, B, and C
    float *A, *B, *C;
    float *d_A, *d_B, *d_C; // Device copies of A, B, C

    A = (float *)malloc(size);
    B = (float *)malloc(size);
    C = (float *)malloc(size);

    // Initialize A and B with some values
    for (int i = 0; i < WIDTH * WIDTH; i++) {
        A[i] = 1.0f; // Some value
        B[i] = 2.0f; // Some value
    }

    // Allocate space for device copies of A, B, C
    checkCudaErrors(hipMalloc((void **)&d_A, size));
    checkCudaErrors(hipMalloc((void **)&d_B, size));
    checkCudaErrors(hipMalloc((void **)&d_C, size));

    // Copy inputs to device
    checkCudaErrors(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));

    // Create events for timing
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

    // Record the start event
    checkCudaErrors(hipEventRecord(start, NULL));


    // Launch matmul_kernel() on the GPU
    dim3 dimBlock(block_size_x, block_size_y);
    dim3 dimGrid((WIDTH + block_size_x * tile_size_x - 1) / (block_size_x * tile_size_x),
             (WIDTH + block_size_y * tile_size_y - 1) / (block_size_y * tile_size_y));
    matmul_kernel<<<dimGrid, dimBlock>>>(d_C, d_A, d_B);

     // Record the stop event
    checkCudaErrors(hipEventRecord(stop, NULL));
    checkCudaErrors(hipEventSynchronize(stop));

    // Check for any errors launching the kernel
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Copy result back to host
    checkCudaErrors(hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost));

    // Calculate the elapsed time in milliseconds
    float milliseconds = 0;
    checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));

    // Compute and print the performance
    double flops = 2.0 * static_cast<double>(WIDTH) * static_cast<double>(WIDTH) * static_cast<double>(WIDTH);
    double gflops = flops / (milliseconds / 1000.0) / 1e9;
    std::cout << "Performance: " << gflops << " GFLOPs" << std::endl;

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Cleanup
    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(A); free(B); free(C);

    return 0;
}

