#include "hip/hip_runtime.h"
#if PRECISION == 32
    #define floatType float
#elif PRECISION == 64
    #define floatType double
#endif

#include "cudacommon.h"
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <iostream>
#include <fstream>
#include "PMSMemMgmt.h"
#include "string.h"

extern "C" {

#include "spmv_kernel_no_template.cu"

using namespace std;

static const int WARP_SIZE = 32;

// Constants

// threshold for error in GPU results
static const double MAX_RELATIVE_ERROR = .02;

// alignment factor in terms of number of floats, used to enforce
// memory coalescing
static const int PAD_FACTOR = 16;

// size of atts buffer
static const int TEMP_BUFFER_SIZE = 1024;

// length of array for reading fields of mtx header
static const int FIELD_LENGTH = 128;

// If using a matrix market pattern, assign values from 0-MAX_RANDOM_VAL
static const float MAX_RANDOM_VAL = 10.0f;

struct Coordinate {
    int x;
    int y;
    float val;
};

void fill(floatType *A, const int n, const float maxi);
void initRandomMatrix(int *cols, int *rowDelimiters, const int n, const int dim);
void convertToColMajor(floatType *A, int *cols, int dim, int *rowDelimiters,
                       floatType *newA, int *newcols, int *rl, int maxrl,
                       bool padded);
void convertToPadded(floatType *A, int *cols, int dim, int *rowDelimiters,
                     floatType **newA_ptr, int **newcols_ptr, int *newIndices,
                     int *newSize);

// ****************************************************************************
// Function: spmvCpu
//
// Purpose:
//   Runs sparse matrix vector multiplication on the CPU
//
// Arguements:
//   val: array holding the non-zero values for the matrix
//   cols: array of column indices for each element of A
//   rowDelimiters: array of size dim+1 holding indices to rows of A;
//                  last element is the index one past the last
//                  element of A
//   vec: dense vector of size dim to be used for multiplication
//   dim: number of rows/columns in the matrix
//   out: input - buffer of size dim
//        output - result from the spmv calculation
//
// Programmer: Lukasz Wesolowski
// Creation: June 23, 2010
// Returns:
//   nothing directly
//   out indirectly through a pointer
// ****************************************************************************
void spmvCpu(const floatType *val, const int *cols, const int *rowDelimiters,
	     const floatType *vec, int dim, floatType *out)
{
    for (int i=0; i<dim; i++)
    {
        floatType t = 0;
        for (int j = rowDelimiters[i]; j < rowDelimiters[i + 1]; j++)
        {
            int col = cols[j];
            t += val[j] * vec[col];
        }
        out[i] = t;
    }
}

// ****************************************************************************
// Function: verifyResults
//
// Purpose:
//   Verifies correctness of GPU results by comparing to CPU results
//
// Arguments:
//   cpuResults: array holding the CPU result vector
//   gpuResults: array hodling the GPU result vector
//   size: number of elements per vector
//   pass: optional iteration number
//
// Programmer: Lukasz Wesolowski
// Creation: June 23, 2010
// Returns:
//   nothing
//   prints "Passed" if the vectors agree within a relative error of
//   MAX_RELATIVE_ERROR and "FAILED" if they are different
// ****************************************************************************
bool verifyResults(const floatType *cpuResults, const floatType *gpuResults,
                   const int size, const int pass = -1)
{
    bool passed = true;
    for (int i = 0; i < size; i++)
    {
        if (fabs(cpuResults[i] - gpuResults[i]) / cpuResults[i]
            > MAX_RELATIVE_ERROR)
        {
//            cout << "Mismatch at i: "<< i << " ref: " << cpuResults[i] <<
//                " dev: " << gpuResults[i] << endl;
            passed = false;
            cerr << "Error: incorrect computed result." << endl;
        }
    }
    if (pass != -1)
    {
        //cout << "Pass "<<pass<<": ";
    }
    if (passed)
    {
        //cout << "Passed" << endl;
    }
    else
    {
        cout << "---FAILED---" << endl;
        cerr << "Error: incorrect computed result." << endl;
    }
    return passed;
}

float csrTest(floatType* h_val,
        int* h_cols, int* h_rowDelimiters, floatType* h_vec, floatType* h_out,
        int numRows, int numNonZeroes, floatType* refOut, bool padded)
{
    // Device data structures
    floatType *d_val, *d_vec, *d_out;
    int *d_cols, *d_rowDelimiters;

    // Allocate device memory
    CUDA_SAFE_CALL(hipMalloc(&d_val,  numNonZeroes * sizeof(floatType)));
    CUDA_SAFE_CALL(hipMalloc(&d_cols, numNonZeroes * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc(&d_vec,  numRows * sizeof(floatType)));
    CUDA_SAFE_CALL(hipMalloc(&d_out,  numRows * sizeof(floatType)));
    CUDA_SAFE_CALL(hipMalloc(&d_rowDelimiters, (numRows+1) * sizeof(int)));


    CUDA_SAFE_CALL(hipMemcpy(d_val, h_val,   numNonZeroes * sizeof(floatType),
            hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_cols, h_cols, numNonZeroes * sizeof(int),
            hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_vec, h_vec, numRows * sizeof(floatType),
                hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_rowDelimiters, h_rowDelimiters,
            (numRows+1) * sizeof(int), hipMemcpyHostToDevice));

    // Bind texture for position
    string suffix;
    if (sizeof(floatType) == sizeof(float))
    {
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
        CUDA_SAFE_CALL(hipBindTexture(0, vecTex, d_vec, channelDesc,
                numRows * sizeof(float)));
        suffix = "-SP";
    }
    else {
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
        CUDA_SAFE_CALL(hipBindTexture(0, vecTexD, d_vec, channelDesc,
                numRows * sizeof(int2)));
        suffix = "-DP";
    }

    int passes = 10;
    int iters  = 100;
    
    // Results description info
    char atts[TEMP_BUFFER_SIZE];
    sprintf(atts, "%d_elements_%d_rows", numNonZeroes, numRows);
    string prefix = "";
    prefix += (padded) ? "Padded_" : "";
    double gflop = 2 * (double) numNonZeroes / 1e9;
    
    // Initialize timers
    hipEvent_t start, stop;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));
    float totalElapsedTime = 0.0;

    // 0: ellpackr, 1: csr-normal-scalar, 2: csr-padded-scalar, 3: csr-normal-vector, 4: csr-padded-vector
    #if (FORMAT == 1 || FORMAT == 2)
        // Setup thread configuration
        int nBlocksScalar = (int) ceil((floatType) numRows / BLOCK_SIZE);
        
        cout << "CSR Scalar Kernel\n";
        for (int k=0; k<passes; k++)
        {
            // Start the timing
            hipEventRecord(start, 0);
            // Run Scalar Kernel
            for (int j = 0; j < iters; j++)
            {
                spmv_csr_scalar_kernel
                <<<nBlocksScalar, BLOCK_SIZE>>>
                (d_val, d_cols, d_rowDelimiters, 
                    #if TEXTURE_MEMORY == 0
                    d_vec,
                    #endif
                    numRows, d_out);
            }

            // Stop the events and save elapsed time
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            float elapsedTime;
            hipEventElapsedTime(&elapsedTime, start, stop);
            totalElapsedTime += elapsedTime;

            CUDA_SAFE_CALL(hipMemcpy(h_out, d_out, numRows * sizeof(floatType),
            hipMemcpyDeviceToHost));
         
            // Compare reference solution to GPU result
            if (! verifyResults(refOut, h_out, numRows, k))
            {
                throw "Results does not match";
                return 100000.0;  // If results don't match, don't report performance
            }
        }
    #else // FORMAT == 3 || FORMAT == 4
        // Setup thread configuration
        int new_block_size = 0;
        if (BLOCK_SIZE < 32) {
            new_block_size = 32;
        } else {
            new_block_size = (int) (ceil((double) BLOCK_SIZE / 32.0) * 32.0);
        }

        int nBlocksVector = (int) ceil((floatType) numRows / (floatType)(new_block_size / WARP_SIZE));

        cout << "CSR Vector Kernel\n";
        for (int k=0; k<passes; k++) {
        
            // Start the timing
            hipEventRecord(start, 0);
            // Run Vector Kernel
            for (int j = 0; j < iters; j++)
            {
                spmv_csr_vector_kernel
                <<<nBlocksVector, new_block_size>>>
                (d_val, d_cols, d_rowDelimiters, 
                    #if TEXTURE_MEMORY == 0
                    d_vec,
                    #endif
                    numRows, d_out);
            }
            
            // Stop the events and save elapsed time
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            float elapsedTime;
            hipEventElapsedTime(&elapsedTime, start, stop);
            totalElapsedTime += elapsedTime;

            CUDA_SAFE_CALL(hipMemcpy(h_out, d_out, numRows * sizeof(floatType),
                    hipMemcpyDeviceToHost));
            hipDeviceSynchronize();
            // Compare reference solution to GPU result
            if (! verifyResults(refOut, h_out, numRows, k))
            {
                throw "Results does not match";
                return 100000.0;  // If results don't match, don't report performance
            }
        }
    #endif
    // Free device memory
    CUDA_SAFE_CALL(hipFree(d_rowDelimiters));
    CUDA_SAFE_CALL(hipFree(d_vec));
    CUDA_SAFE_CALL(hipFree(d_out));
    CUDA_SAFE_CALL(hipFree(d_val));
    CUDA_SAFE_CALL(hipFree(d_cols));
    CUDA_SAFE_CALL(hipUnbindTexture(vecTexD));
    CUDA_SAFE_CALL(hipUnbindTexture(vecTex));
    CUDA_SAFE_CALL(hipEventDestroy(start));
    CUDA_SAFE_CALL(hipEventDestroy(stop));
    return totalElapsedTime;
}

float ellPackTest(floatType* h_val,
        int* h_cols, int* h_rowDelimiters, floatType* h_vec, floatType* h_out,
        int numRows, int numNonZeroes, floatType* refOut, bool padded,
        int paddedSize)
{
    int *h_rowLengths;
    CUDA_SAFE_CALL(hipHostMalloc(&h_rowLengths, paddedSize * sizeof(int)));
    int maxrl = 0;
    for (int k=0; k<numRows; k++)
    {
        h_rowLengths[k] = h_rowDelimiters[k+1] - h_rowDelimiters[k];
        if (h_rowLengths[k] > maxrl)
        {
            maxrl = h_rowLengths[k];
        }
    }
    for (int p=numRows; p < paddedSize; p++)
    {
        h_rowLengths[p] = 0;
    }

    // Column major format host data structures
    int cmSize = padded ? paddedSize : numRows;
    floatType *h_valcm;
    CUDA_SAFE_CALL(hipHostMalloc(&h_valcm, maxrl * cmSize * sizeof(floatType)));
    int *h_colscm;
    CUDA_SAFE_CALL(hipHostMalloc(&h_colscm, maxrl * cmSize * sizeof(int)));
    convertToColMajor(h_val, h_cols, numRows, h_rowDelimiters, h_valcm,
                              h_colscm, h_rowLengths, maxrl, padded);

    // Device data structures
    floatType *d_val, *d_vec, *d_out;
    int *d_cols, *d_rowLengths;

    // Allocate device memory
    CUDA_SAFE_CALL(hipMalloc(&d_val,  maxrl*cmSize * sizeof(floatType)));
    CUDA_SAFE_CALL(hipMalloc(&d_cols, maxrl*cmSize * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc(&d_vec,  numRows * sizeof(floatType)));
    CUDA_SAFE_CALL(hipMalloc(&d_out,  paddedSize * sizeof(floatType)));
    CUDA_SAFE_CALL(hipMalloc(&d_rowLengths, cmSize * sizeof(int)));

    // Transfer data to device
    CUDA_SAFE_CALL(hipMemcpy(d_val, h_valcm, maxrl*cmSize * sizeof(floatType),
            hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_cols, h_colscm, maxrl*cmSize * sizeof(int),
            hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_vec, h_vec, numRows * sizeof(floatType),
            hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(d_rowLengths, h_rowLengths,
            cmSize * sizeof(int), hipMemcpyHostToDevice));

    // Bind texture for position
    if (sizeof(floatType) == sizeof(float))
    {
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
        CUDA_SAFE_CALL(hipBindTexture(0, vecTex, d_vec, channelDesc,
                numRows * sizeof(float)));
    }
    else
    {
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
        CUDA_SAFE_CALL(hipBindTexture(0, vecTexD, d_vec, channelDesc,
                numRows * sizeof(int2)));
    }
    int nBlocks = (int) ceil((floatType) cmSize / BLOCK_SIZE);
    int passes = 10;
    int iters  = 100;

    // Initialize timers
    hipEvent_t start, stop;
    CUDA_SAFE_CALL(hipEventCreate(&start));
    CUDA_SAFE_CALL(hipEventCreate(&stop));
    float totalElapsedTime = 0.0;

    for (int k=0; k<passes; k++) {
        
        // Start the timing
        hipEventRecord(start, 0);

        for (int j = 0; j < iters; j++) {
            spmv_ellpackr_kernel<<<nBlocks, BLOCK_SIZE>>>
                    (d_val, d_cols, d_rowLengths,
                        #if TEXTURE_MEMORY == 0
                        d_vec,
                        #endif
                        cmSize, d_out);
        }

        // Stop the events and save elapsed time
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, start, stop);
        totalElapsedTime += elapsedTime;

        CUDA_SAFE_CALL(hipMemcpy(h_out, d_out, cmSize * sizeof(floatType),
                hipMemcpyDeviceToHost));

        // Compare reference solution to GPU result
        if (! verifyResults(refOut, h_out, numRows, k)) {
            throw "Results does not match";
            return 100000.0;  // If results don't match, don't report performance
        }
    }

    // Free device memory
    CUDA_SAFE_CALL(hipFree(d_rowLengths));
    CUDA_SAFE_CALL(hipFree(d_vec));
    CUDA_SAFE_CALL(hipFree(d_out));
    CUDA_SAFE_CALL(hipFree(d_val));
    CUDA_SAFE_CALL(hipFree(d_cols));
    if (sizeof(floatType) == sizeof(double))
    {
        CUDA_SAFE_CALL(hipUnbindTexture(vecTexD));
    }
    else
    {
        CUDA_SAFE_CALL(hipUnbindTexture(vecTex));
    }
    CUDA_SAFE_CALL(hipHostFree(h_rowLengths));
    CUDA_SAFE_CALL(hipHostFree(h_valcm));
    CUDA_SAFE_CALL(hipHostFree(h_colscm));
    CUDA_SAFE_CALL(hipEventDestroy(start));
    CUDA_SAFE_CALL(hipEventDestroy(stop));
    return totalElapsedTime;
}

// ****************************************************************************
// Function: RunTest
//
// Purpose:
//   Executes a run of the sparse matrix - vector multiplication benchmark
//   in either single or double precision
//
// Arguments:
//   op: the options parser / parameter database
//   nRows: number of rows in generated matrix
//
// Returns:  nothing
//
// Programmer: Lukasz Wesolowski
// Creation: June 21, 2010
//
// Modifications:
//
// ****************************************************************************
float RunTest(int nRows=0)
{
    // Host data structures
    // Array of values in the sparse matrix
    floatType *h_val, *h_valPad;
    // Array of column indices for each value in h_val
    int *h_cols, *h_colsPad;
    // Array of indices to the start of each row in h_Val
    int *h_rowDelimiters, *h_rowDelimitersPad;
    // Dense vector and space for dev/cpu reference solution
    floatType *h_vec, *h_out, *refOut;
    // nItems = number of non zero elems
    int nItems, nItemsPadded, numRows;

    // This generates a random matrix
    numRows = nRows;
    nItems = numRows * numRows / 100; // 1% of entries will be non-zero
    float maxval = 10;
    CUDA_SAFE_CALL(hipHostMalloc(&h_val, nItems * sizeof(floatType)));
    CUDA_SAFE_CALL(hipHostMalloc(&h_cols, nItems * sizeof(int)));
    CUDA_SAFE_CALL(hipHostMalloc(&h_rowDelimiters, (numRows + 1) * sizeof(int)));
    fill(h_val, nItems, maxval);
    initRandomMatrix(h_cols, h_rowDelimiters, nItems, numRows);

    // Set up remaining host data
    CUDA_SAFE_CALL(hipHostMalloc(&h_vec, numRows * sizeof(floatType)));
    refOut = new floatType[numRows];
    CUDA_SAFE_CALL(hipHostMalloc(&h_rowDelimitersPad, (numRows + 1) * sizeof(int)));
    fill(h_vec, numRows, 10);

    // Set up the padded data structures
    int paddedSize = numRows + (PAD_FACTOR - numRows % PAD_FACTOR);
    CUDA_SAFE_CALL(hipHostMalloc(&h_out, paddedSize * sizeof(floatType)));
    convertToPadded(h_val, h_cols, numRows, h_rowDelimiters, &h_valPad,
            &h_colsPad, h_rowDelimitersPad, &nItemsPadded);

    // Compute reference solution
    spmvCpu(h_val, h_cols, h_rowDelimiters, h_vec, numRows, refOut);

    float result = 0.0;
    // 0: ellpackr, 1: csr-normal-scalar, 2: csr-padded-scalar, 3: csr-normal-vector, 4: csr-padded-vector
    #if (FORMAT == 1 || FORMAT == 3)
        // Test CSR kernels on normal data
        cout << "CSR Test\n";
        result = csrTest(h_val, h_cols,
                h_rowDelimiters, h_vec, h_out, numRows, nItems, refOut, false);
    #elif (FORMAT == 2 || FORMAT == 4)
        // Test CSR kernels on padded data
        cout << "CSR Test -- Padded Data\n";
        result = csrTest(h_valPad, h_colsPad,
                h_rowDelimitersPad, h_vec, h_out, numRows, nItemsPadded, refOut, true);
    #else
        // FORMAT == 0
        // Test ELLPACKR kernel
        cout << "ELLPACKR Test\n";
        result = ellPackTest(h_val, h_cols,
                h_rowDelimiters, h_vec, h_out, numRows, nItems, refOut, false,
                paddedSize);
    #endif

    delete[] refOut;
    CUDA_SAFE_CALL(hipHostFree(h_val));
    CUDA_SAFE_CALL(hipHostFree(h_cols));
    CUDA_SAFE_CALL(hipHostFree(h_rowDelimiters));
    CUDA_SAFE_CALL(hipHostFree(h_vec));
    CUDA_SAFE_CALL(hipHostFree(h_out));
    CUDA_SAFE_CALL(hipHostFree(h_valPad));
    CUDA_SAFE_CALL(hipHostFree(h_colsPad));
    CUDA_SAFE_CALL(hipHostFree(h_rowDelimitersPad));

    return result;
}

// ****************************************************************************
// Function: RunBenchmark
//
// Purpose:
//   Executes the sparse matrix - vector multiplication benchmark
//
// Arguments:
//   resultDB: stores results from the benchmark
//   op: the options parser / parameter database
//
// Returns:  nothing
//
// Programmer: Lukasz Wesolowski
// Creation: June 21, 2010
//
// Modifications:
//
// ****************************************************************************
float RunBenchmark()
{
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    int probSizes[4] = {1024, 8192, 12288, 16384};
    int sizeClass = PROBLEM_SIZE - 1;

    return RunTest(probSizes[sizeClass]);
}

// Spmv/util.h 

void fill(floatType *A, const int n, const float maxi)
{
    for (int j = 0; j < n; j++)
    {
        A[j] = ((floatType) maxi * (rand() / (RAND_MAX + 1.0f)));
    }
}

void convertToColMajor(floatType *A, int *cols, int dim, int *rowDelimiters,
    floatType *newA, int *newcols, int *rl, int maxrl,
    bool padded)
{
    int pad = 0;
    if (padded && dim % PAD_FACTOR != 0) {
        pad = PAD_FACTOR - dim % PAD_FACTOR;
    }

    int newIndex = 0;
    for (int j=0; j<maxrl; j++) {
        for (int i=0; i<dim; i++) {
            if (rowDelimiters[i] + j < rowDelimiters[i+1]) {
                newA[newIndex] = A[rowDelimiters[i]+j];
                newcols[newIndex] = cols[rowDelimiters[i]+j];
            } else {
                newA[newIndex] = 0;
            }
            newIndex++;
        }
        if (padded) {
            for (int p=0; p<pad; p++) {
                newA[newIndex] = 0;
                newIndex++;
            }
        }
    }
}


void convertToPadded(floatType *A, int *cols, int dim, int *rowDelimiters,
                     floatType **newA_ptr, int **newcols_ptr, int *newIndices,
                     int *newSize)
{

    // determine total padded size and new row indices
    int paddedSize = 0;
    int rowSize;

    for (int i=0; i<dim; i++)
    {
        newIndices[i] = paddedSize;
        rowSize = rowDelimiters[i+1] - rowDelimiters[i];
        if (rowSize % PAD_FACTOR != 0)
        {
            rowSize += PAD_FACTOR - rowSize % PAD_FACTOR;
        }
        paddedSize += rowSize;
    }
    *newSize = paddedSize;
    newIndices[dim] = paddedSize;

    *newA_ptr = pmsAllocHostBuffer<floatType>( paddedSize );
    *newcols_ptr = pmsAllocHostBuffer<int>( paddedSize );

    floatType *newA = *newA_ptr;
    int *newcols = *newcols_ptr;

    memset(newA, 0, paddedSize * sizeof(floatType));
    memset(newcols, 0, paddedSize*sizeof(int));

    // fill newA and newcols
    for (int i=0; i<dim; i++)
    {
        for (int j=rowDelimiters[i], k=newIndices[i]; j<rowDelimiters[i+1];
             j++, k++)
        {
            newA[k] = A[j];
            newcols[k] = cols[j];
        }
    }
}

void initRandomMatrix(int *cols, int *rowDelimiters, const int n, const int dim)
{
    int nnzAssigned = 0;

    // Figure out the probability that a nonzero should be assigned to a given
    // spot in the matrix
    double prob = (double)n / ((double)dim * (double)dim);

    // Seed random number generator
    srand48(8675309L);

    // Randomly decide whether entry i,j gets a value, but ensure n values
    // are assigned
    bool fillRemaining = false;
    for (int i = 0; i < dim; i++)
    {
        rowDelimiters[i] = nnzAssigned;
        for (int j = 0; j < dim; j++)
        {
            int numEntriesLeft = (dim * dim) - ((i * dim) + j);
            int needToAssign   = n - nnzAssigned;
            if (numEntriesLeft <= needToAssign) {
                fillRemaining = true;
            }
            if ((nnzAssigned < n && drand48() <= prob) || fillRemaining)
            {
                // Assign (i,j) a value
                cols[nnzAssigned] = j;
                nnzAssigned++;
            }
        }
    }
    // Observe the convention to put the number of non zeroes at the end of the
    // row delimiters array
    rowDelimiters[dim] = n;
    assert(nnzAssigned == n);
}

}
